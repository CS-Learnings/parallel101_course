#include <hip/hip_runtime.h>
#include <glm/glm.hpp>
#include <glm/ext.hpp>
#include <vector>

#define CHECK_CUDA(expr) do { \
    hipError_t err = (expr); \
    if (err != hipSuccess) { \
        fprintf(stderr, "%s:%d: %s: %s\n", \
                __FILE__, __LINE__, \
                #expr, hipGetErrorString(err)); \
        abort(); \
    } \
} while (0)

// 用 CUDA 实现分子动力学吧！

struct Atom {
    glm::vec3 r;
    glm::vec3 v;
    glm::vec3 F;
    float m;
};

std::vector<Atom> atoms;

int main() {
    float dt = 0.01f;
    for (auto &atom: atoms) {
        atom.v += atom.F / atom.m * dt / 2.0f;
    }
    for (auto &atom: atoms) {
        atom.r += atom.v * dt;
    }
    for (auto &atom: atoms) {
        atom.F = {};
    }
    for (auto &atom: atoms) {
        atom.v += atom.F / atom.m * dt / 2.0f;
    }
}
